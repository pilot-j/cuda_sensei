
#include <hip/hip_runtime.h>
#include <iostream>
#define N 10

//kernel to add vectors larger than the actual number of threads * blocks
__global__ void vec_add(int *d_mat1, int *d_mat2, int *d_out, int n){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < n){
        d_out[tid] = d_mat1[tid] + d_mat2[tid];
        int offset = blockDim.x * gridDim.x;
        tid += offset;
    }
}

/*
//basic kernel with parallelisation across threads
__global__ void vec_add(int *d_v1, int *d_v2, int *d_out, int n ) {
    int tid = threadIdx.x;

    if(tid<n){
        d_out[tid]= d_v1[tid] + d_v2[tid];
    }
}
*/

int main(void){
    int h_vec1[N], h_vec2[N], h_out[N];
    int *d_vec1, *d_vec2, *d_out;
    
    dim3 grid_size(1); // we take 1 blocks with N threads
    dim3 block_size(N);

    /*Alternate way to launch kernels
    int threads = x
    dim3 block_size(x);
    dim3 grid_size((N + threads - 1) / threads); 
    */

    for(int i =0;i<N;i++){
        h_vec1[i]= 2*i;
        h_vec2[i]= i;
    }
    //in usual cases we will receive these host arrays from somewhere

    hipMalloc((void**)&d_vec1, N*sizeof(int));
    hipMalloc((void**)&d_vec2, N*sizeof(int));
    hipMalloc((void**)&d_out,  N*sizeof(int));

    hipMemcpy(d_vec1, h_vec1,N*sizeof(int) ,hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, h_vec2,N*sizeof(int) ,hipMemcpyHostToDevice);
    
    vec_add<<<grid_size, block_size>>>(d_vec1, d_vec2, d_out, N);

    hipMemcpy(h_out,  d_out ,N*sizeof(int) ,hipMemcpyHostToDevice);

    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_out);
     
    for(int i =0;i<N;i++){
        printf("%d + %d = %d \n", h_vec1[i], h_vec2[i], h_out[i]);
    }

    return 0;
}
